#include "Pbf.h"
#include "PbfImpl.cuh"

Pbf::Pbf(float radius, float rho0, float eps, int maxIter, float c, float kCorr)
    : mImpl(new PbfImpl(radius, rho0, eps, maxIter, c, kCorr))
{
}

void Pbf::setParticles(const std::vector<Particle>& particles)
{
    mImpl->setParticles(particles);
}

const std::vector<Particle>& Pbf::getParticles() const
{
    return mImpl->getParticles();
}

void Pbf::step(float dt)
{
    mImpl->step(dt);
}
