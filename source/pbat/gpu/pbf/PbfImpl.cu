#include "hip/hip_runtime.h"
#include "PbfImpl.cuh"
#include "PbfKernels.cuh"
#include "common/CUDAUtils.h"

Pbf::PbfImpl::PbfImpl(float radius_, float rho0_, float eps_, int maxIter_, float c_, float kCorr_)
    : radius(radius_), rho0(rho0_), eps(eps_), maxIter(maxIter_), c(c_), kCorr(kCorr_),
      hashGrid(make_float3(-2.0f, 0.0f, -1.0f), make_float3(2.0f, 2.0f, 1.0f), radius)
{
}

void Pbf::PbfImpl::setParticles(const std::vector<Particle>& particles)
{
    h_particles = particles;
    const size_t numParticles = h_particles.size();

    d_positions.resize(numParticles);
    d_velocities.resize(numParticles);
    d_predictedPositions.resize(numParticles);
    d_densities.resize(numParticles);
    d_lambdas.resize(numParticles);
    d_keys.resize(numParticles);

    thrust::host_vector<float3> h_positions(numParticles);
    thrust::host_vector<float3> h_velocities(numParticles);
    for (size_t i = 0; i < numParticles; ++i)
    {
        h_positions[i] = make_float3(h_particles[i].x.x(), h_particles[i].x.y(), h_particles[i].x.z());
        h_velocities[i] = make_float3(h_particles[i].v.x(), h_particles[i].v.y(), h_particles[i].v.z());
    }

    d_positions = h_positions;
    d_velocities = h_velocities;
}

const std::vector<Particle>& Pbf::PbfImpl::getParticles() const
{
    return h_particles;
}

void Pbf::PbfImpl::step(float dt)
{
    updatePredictedPositions(dt);
    updateHashGrid();
    buildNeighborhood();
    for (int iter = 0; iter < maxIter; ++iter)
    {
        computeDensityAndLambda();
        computeDeltaP();
        boxCollision();
    }
    updatePositionsAndVelocities(dt);
    applyViscosity();
    applyVorticityConfinement();
}

void Pbf::PbfImpl::updatePredictedPositions(float dt)
{
    const size_t numParticles = h_particles.size();
    const float3 externalForce = make_float3(0.0f, -9.8f, 0.0f);

    PbfKernels::updatePredictedPositions<<<(numParticles + 255) / 256, 256>>>(
        thrust::raw_pointer_cast(d_positions.data()),
        thrust::raw_pointer_cast(d_velocities.data()),
        thrust::raw_pointer_cast(d_predictedPositions.data()),
        externalForce, dt, numParticles);
}

void Pbf::PbfImpl::updateHashGrid()
{
    // Implement hash grid update using CUDA if needed
}

void Pbf::PbfImpl::buildNeighborhood()
{
    // Implement neighborhood building using CUDA if needed
}

void Pbf::PbfImpl::computeDensityAndLambda()
{
    const size_t numParticles = h_particles.size();
    Kernel kernel(radius);

    PbfKernels::computeDensityAndLambda<<<(numParticles + 255) / 256, 256>>>(
        thrust::raw_pointer_cast(d_predictedPositions.data()),
        thrust::raw_pointer_cast(d_densities.data()),
        thrust::raw_pointer_cast(d_lambdas.data()),
        rho0, eps, numParticles, kernel);
}

void Pbf::PbfImpl::computeDeltaP()
{
    const size_t numParticles = h_particles.size();
    Kernel kernel(radius);

    PbfKernels::computeDeltaP<<<(numParticles + 255) / 256, 256>>>(
        thrust::raw_pointer_cast(d_predictedPositions.data()),
        thrust::raw_pointer_cast(d_lambdas.data()),
        thrust::raw_pointer_cast(d_predictedPositions.data()),
        rho0, kCorr, numParticles, kernel);
}

void Pbf::PbfImpl::updatePositionsAndVelocities(float dt)
{
    const size_t numParticles = h_particles.size();

    PbfKernels::updatePositionsAndVelocities<<<(numParticles + 255) / 256, 256>>>(
        thrust::raw_pointer_cast(d_positions.data()),
        thrust::raw_pointer_cast(d_velocities.data()),
        thrust::raw_pointer_cast(d_predictedPositions.data()),
        dt, numParticles);
}

void Pbf::PbfImpl::applyViscosity()
{
    // Implement viscosity application using CUDA if needed
}

void Pbf::PbfImpl::applyVorticityConfinement()
{
    // Implement vorticity confinement using CUDA if needed
}

void Pbf::PbfImpl::boxCollision()
{
    const size_t numParticles = h_particles.size();
    const float3 min = make_float3(-2.0f, 0.0f, -1.0f);
    const float3 max = make_float3(2.0f, 2.0f, 1.0f);

    PbfKernels::boxCollision<<<(numParticles + 255) / 256, 256>>>(
        thrust::raw_pointer_cast(d_predictedPositions.data()),
        min, max, numParticles);
}
