#include "hip/hip_runtime.h"
#include "KernelImpl.cuh"
#include <algorithm> // For std::max

namespace SPH
{
    float poly6_kernel(const Eigen::Vector3f& pi, const Eigen::Vector3f& pj, float h2, float kPoly)
    {
        Eigen::Vector3f r_vec = pi - pj;
        float r2 = r_vec.squaredNorm();
        if (r2 < h2)
        {
            float x = h2 - r2;
            return kPoly * x * x * x;
        }
        return 0.0f;
    }

    Eigen::Vector3f spiky_kernel(const Eigen::Vector3f& pi, const Eigen::Vector3f& pj, float h, float kSpiky)
    {
        Eigen::Vector3f r_vec = pi - pj;
        float r = r_vec.norm();

        if (r < 1e-6f)
        {
            return Eigen::Vector3f::Zero();
        }

        if (r < h)
        {
            float x = h - r;
            float coeff = (kSpiky * x * x) / r;
            return -coeff * r_vec;
        }

        return Eigen::Vector3f::Zero();
    }
} // namespace SPH
